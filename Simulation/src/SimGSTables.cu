#include "hip/hip_runtime.h"
#include "SimGSTables.hh"

#include <iostream>
#include <cstdio>
#include <cmath>
#include "Utils.h"

namespace GSTables {
   __constant__ int   SamplingTableSize;
   __constant__ float MinPrimaryEnergy;
   __constant__ float LogMinPrimaryEnergy;
   __constant__ float InvLogDeltaPrimaryEnergy;
   __constant__ float DeltaCum;

   hipArray_t arrVarU;
   hipArray_t arrParaA;
   hipArray_t arrParaB;
   hipArray_t arrTransformParam;
   hipArray_t arrPrimaryEnergyGrid;

   hipTextureObject_t texVarU, texParaA, texParaB, texTransformParam, texPrimaryEnergyGrid;
   __device__ hipTextureObject_t d_texVarU, d_texParaA, d_texParaB, d_texTransformParam, d_texPrimaryEnergyGrid;

    __device__ float SampleAngularDeflection(float eprim, float rndm1, float rndm2) {
        // determine electron energy lower grid point and sample if that or one above is used now
        float lpenergy = std::log(eprim);
        float phigher = (lpenergy - LogMinPrimaryEnergy) * InvLogDeltaPrimaryEnergy;
        int penergyindx = (int)phigher;
        // keep the lower index of the energy bin
        const int ielow = penergyindx;
        phigher -= penergyindx;
        if (rndm1 < phigher) {
            ++penergyindx;
        }
        // should always be fine if electron-cut < eprim < E_max but make sure
      //  penergyindx      = std::min(fNumPrimaryEnergies-1, penergyindx);
        // sample the transformed variable \xi

        // lower index of the (common) discrete cumulative bin and the residual fraction
        const int    indxl = (int)(rndm2 / DeltaCum);
        const float resid = rndm2 - indxl * DeltaCum;
        // compute value `u` by using ratin based numerical inversion
        const float  parA = tex2D<float>(d_texParaA, indxl+0.5f, penergyindx + 0.5f);
        const float  parB = tex2D<float>(d_texParaB, indxl + 0.5f, penergyindx + 0.5f);
        const float    u0 = tex2D<float>(d_texVarU, indxl + 0.5f, penergyindx + 0.5f);
        const float    u1 = tex2D<float>(d_texVarU, indxl+1 + 0.5f, penergyindx + 0.5f);
        const float  dum1 = (1.0f + parA + parB) * DeltaCum * resid;
        const float  dum2 = DeltaCum * DeltaCum + parA * DeltaCum * resid + parB * resid * resid;
        const float  theU = u0 + dum1 / dum2 * (u1 - u0);

        // transform back the sampled `u` to `mu(u)` using the transformation parameter `a`
        // mu(u) = 1 - 2au/[1-u+a] as given by Eq.(34)
        // interpolate (linearly) the transformation parameter to E
        const float a0 = tex1D<float>(d_texTransformParam, ielow + 0.5f);
        const float a1 = tex1D<float>(d_texTransformParam, ielow+1 + 0.5f);
        const float e0 = tex1D<float>(d_texPrimaryEnergyGrid, ielow + 0.5f);
        const float e1 = tex1D<float>(d_texPrimaryEnergyGrid, ielow+1 + 0.5f);

        const float parTransf = (a1 - a0) / (e1 - e0) * (eprim - e0) + a0;       
        return 1.f - 2.f * parTransf * theU / (1.f - theU + parTransf);
    }
}


void SimGSTables::InitializeTables()
{
    float auxilary;
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::SamplingTableSize), &fSamplingTableSize, sizeof(int));
    auxilary = (float)fMinPrimaryEnergy;
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::MinPrimaryEnergy), &auxilary, sizeof(float));
    auxilary = (float)fLogMinPrimaryEnergy;
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::LogMinPrimaryEnergy), &auxilary, sizeof(float));
    auxilary = (float)fInvLogDeltaPrimaryEnergy;
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::InvLogDeltaPrimaryEnergy), &auxilary, sizeof(float));
	auxilary = (float)fDeltaCum;
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::DeltaCum), &auxilary, sizeof(float));


    std::vector<float> VarUTable;
    std::vector<float> ParaATable;
    std::vector<float> ParaBTable;
    std::vector<float> TransformParamTable;
    std::vector<float> PrimaryEnergyGridTable;

    TransformParamTable.resize(fNumPrimaryEnergies);
    PrimaryEnergyGridTable.resize(fNumPrimaryEnergies);
    VarUTable.resize(fNumPrimaryEnergies * fSamplingTableSize);
    ParaATable.resize(fNumPrimaryEnergies * fSamplingTableSize);
    ParaBTable.resize(fNumPrimaryEnergies * fSamplingTableSize);
    for (int i = 0; i < fNumPrimaryEnergies; ++i) {
        TransformParamTable[i] = (float)fTheTables[i]->fTransformParam;
        PrimaryEnergyGridTable[i] = (float)fPrimaryEnergyGrid[i];

        for (int j = 0; j < fSamplingTableSize; ++j){
            VarUTable[i * fSamplingTableSize + j] = (float)fTheTables[i]->fGSTable[j].fVarU;
            ParaATable[i * fSamplingTableSize + j] = (float)fTheTables[i]->fGSTable[j].fParmA;
            ParaBTable[i * fSamplingTableSize + j] = (float)fTheTables[i]->fGSTable[j].fParmB;
        }
    }

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.normalizedCoords = 0;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;

    initCudaTexture(TransformParamTable.data(), &fNumPrimaryEnergies, 1, &texDesc, GSTables::texTransformParam, GSTables::arrTransformParam);
    initCudaTexture(PrimaryEnergyGridTable.data(), &fNumPrimaryEnergies, 1, &texDesc, GSTables::texPrimaryEnergyGrid, GSTables::arrPrimaryEnergyGrid);
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texTransformParam), &GSTables::texTransformParam, sizeof(hipTextureObject_t));
    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texPrimaryEnergyGrid), &GSTables::texPrimaryEnergyGrid, sizeof(hipTextureObject_t));

    int size[2] = { fSamplingTableSize, fNumPrimaryEnergies };
	initCudaTexture(VarUTable.data(), size, 2, &texDesc, GSTables::texVarU, GSTables::arrVarU);
	initCudaTexture(ParaATable.data(), size, 2, &texDesc, GSTables::texParaA, GSTables::arrParaA);
	initCudaTexture(ParaBTable.data(), size, 2, &texDesc, GSTables::texParaB, GSTables::arrParaB);

    hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texVarU), &GSTables::texVarU, sizeof(hipTextureObject_t));
	hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texParaA), &GSTables::texParaA, sizeof(hipTextureObject_t));
	hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texParaB), &GSTables::texParaB, sizeof(hipTextureObject_t));
	hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texTransformParam), &GSTables::texTransformParam, sizeof(hipTextureObject_t));
	hipMemcpyToSymbol(HIP_SYMBOL(GSTables::d_texPrimaryEnergyGrid), &GSTables::texPrimaryEnergyGrid, sizeof(hipTextureObject_t));

    CudaCheckError();
}

SimGSTables::SimGSTables() {
  // all members will be set when loading the data from the file
  fSamplingTableSize        = -1;
  fNumPrimaryEnergies       = -1;
  fMinPrimaryEnergy         = -1.;
  fLogMinPrimaryEnergy      = -1.;
  fInvLogDeltaPrimaryEnergy = -1.;
  fDeltaCum                 = -1.;
}

void SimGSTables::LoadData(const std::string& dataDir, int verbose) {
  char name[512];
  sprintf(name, "%s/el_GSDtrData.dat", dataDir.c_str());
  FILE* f = fopen(name, "r");
  if (!f) {
    std::cerr << " *** ERROR SimGSTables::LoadData: \n"
              << "     file = " << name << " not found! "
              << std::endl;
    exit(EXIT_FAILURE);
  }
  // first 4 lines are comments
  for (int i=0; i<4; ++i) { fgets(name, sizeof(name), f); }
  // load the size of the primary energy grid and the individual tables first
  double ddum;
  fscanf(f, "%d %lg %lg %d\n", &fNumPrimaryEnergies, &ddum, &ddum, &fSamplingTableSize);
  if (verbose > 0) {
    std::cout << " == Loading GS tables: "
              << fNumPrimaryEnergies << " tables with a size of "
              << fSamplingTableSize << " each."
              << std::endl;
  }
  fDeltaCum = 1.0/(fSamplingTableSize-1);
  // skipp the next 6 lines that are comments
  for (int i=0; i<6; ++i) {
    fgets(name, sizeof(name), f);
    if (i==4 && verbose>0) {
      std::cout << "    --- The GS angular data were computed for: " << name;
    }
  }
  // clean the tables if any
  CleanTables();
  fTheTables.resize(fNumPrimaryEnergies, nullptr);
  fPrimaryEnergyGrid.resize(fNumPrimaryEnergies);
  // load each primary energies and at each primary energy the corresponding table
  for (int ie=0; ie<fNumPrimaryEnergies; ++ie) {
    // load the table index, primary particle kinetic energy and transformation paramater values
    int    idum;
    double transPar;
    fscanf(f, "%d %lg %lg", &idum, &ddum, &transPar);
    fPrimaryEnergyGrid[ie] = ddum;
    if (ie==0) {
      // this is the electron-cut that is also the e-/e+ absorption energy
      fMinPrimaryEnergy    = ddum;
      fLogMinPrimaryEnergy = std::log(ddum);
    }
    if (ie==1) {
      fInvLogDeltaPrimaryEnergy = 1./(std::log(ddum)-fLogMinPrimaryEnergy);
    }
    // construct a sampling table, load the data and fill in the sampling table
    fTheTables[ie] = new OneGSTable();
    fTheTables[ie]->fTransformParam = transPar;
    fTheTables[ie]->fGSTable.resize(fSamplingTableSize);
    for (int is=0; is<fSamplingTableSize; ++is) {
      double u, a, b;
      fscanf(f, "%lg %lg %lg", &u, &a, &b);
      fTheTables[ie]->fGSTable[is].fVarU  = u;
      fTheTables[ie]->fGSTable[is].fParmA = a;
      fTheTables[ie]->fGSTable[is].fParmB = b;
    }
  }
  fclose(f);

  InitializeTables();
}

// it is assumed that the `eprim` electron energy: electron-cut < eprim <E_max
double SimGSTables::SampleAngularDeflection(double eprim, double rndm1, double rndm2) {
  // determine electron energy lower grid point and sample if that or one above is used now
  double lpenergy  = std::log(eprim);
  double phigher   = (lpenergy-fLogMinPrimaryEnergy)*fInvLogDeltaPrimaryEnergy;
  int penergyindx  = (int) phigher;
  // keep the lower index of the energy bin
  const int ielow  = penergyindx;
  phigher         -= penergyindx;
  if (rndm1<phigher) {
    ++penergyindx;
  }
  // should always be fine if electron-cut < eprim < E_max but make sure
//  penergyindx      = std::min(fNumPrimaryEnergies-1, penergyindx);
  // sample the transformed variable \xi
  const OneGSTable* theGSTable = fTheTables[penergyindx];
  // lower index of the (common) discrete cumulative bin and the residual fraction
  const int    indxl = (int)(rndm2/fDeltaCum);
  const double resid = rndm2-indxl*fDeltaCum;
  // compute value `u` by using ratin based numerical inversion
  const double  parA = theGSTable->fGSTable[indxl].fParmA;
  const double  parB = theGSTable->fGSTable[indxl].fParmB;
  const double    u0 = theGSTable->fGSTable[indxl].fVarU;
  const double    u1 = theGSTable->fGSTable[indxl+1].fVarU;
  const double  dum1 = (1.0 + parA + parB) * fDeltaCum * resid;
  const double  dum2 = fDeltaCum * fDeltaCum + parA * fDeltaCum * resid + parB * resid * resid;
  const double  theU = u0 + dum1 / dum2 * (u1 - u0);
  // transform back the sampled `u` to `mu(u)` using the transformation parameter `a`
  // mu(u) = 1 - 2au/[1-u+a] as given by Eq.(34)
  // interpolate (linearly) the transformation parameter to E
  const double a0 = fTheTables[ielow]->fTransformParam;
  const double a1 = fTheTables[ielow+1]->fTransformParam;
  const double e0 = fPrimaryEnergyGrid[ielow];
  const double e1 = fPrimaryEnergyGrid[ielow+1];
  const double parTransf = (a1-a0)/(e1-e0)*(eprim-e0)+a0;
  return 1.-2.*parTransf*theU/(1.-theU+parTransf);
}


void SimGSTables::CleanTables() {
  for (std::size_t i=0; i<fTheTables.size(); ++i) {
    if (fTheTables[i]) delete fTheTables[i];
  }
  fTheTables.clear();
}
