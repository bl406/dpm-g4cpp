#include "hip/hip_runtime.h"
#include "Random.hh"

std::mt19937 Random::generator(123);
std::uniform_real_distribution<double> Random::dis(0.0,1.0);

namespace CuRand {
    __device__ hiprandState* d_states;

    /* ��ʼ�������� */
    __global__ void initGenerator(unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &d_states[idx]);
    }

    void initCurand(int blocks, int threads) {
        hiprandState* states;
        hipMalloc(&states, blocks * threads * sizeof(hiprandState));		
		hipMemcpyToSymbol(HIP_SYMBOL(d_states), &states, sizeof(hiprandState*));
        initGenerator << <blocks, threads >> > (time(NULL));
    }

    /* ���һ����[0, 1)�����ھ��ȷֲ���double�ͱ��� */
    __device__ float rand() {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        return hiprand_uniform(&d_states[idx]);
    }
}