#include "hip/hip_runtime.h"
#include "Random.hh"

std::mt19937 Random::generator(123);
std::uniform_real_distribution<double> Random::dis(0.0,1.0);

namespace CuRand {
    __device__ hiprandState* d_states;
    float* RandomArray;
    __device__ float* d_RandomArray;

#define N 65536

    /* ��ʼ�������� */
    __global__ void initGenerator(unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &d_states[idx]);
    }

    void initCurand(int blocks, int threads) {
        hiprandState* states;
        hipMalloc(&states, blocks * threads * sizeof(hiprandState));		
		hipMemcpyToSymbol(HIP_SYMBOL(d_states), &states, sizeof(hiprandState*));
        initGenerator << <blocks, threads >> > (time(NULL));

        hipMalloc(&RandomArray, N*sizeof(float));
        std::vector<float> randoms(N);
        for (size_t i = 0; i < randoms.size(); ++i) {
            randoms[i] = Random::UniformRand();
        }
        hipMemcpy(RandomArray, randoms.data(), randoms.size() * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(d_RandomArray), &RandomArray, sizeof(float*));
    }

    /* ���һ����[0, 1)�����ھ��ȷֲ���double�ͱ��� */
    __device__ float rand() {
        /*int idx = blockIdx.x * blockDim.x + threadIdx.x;
        return hiprand_uniform(&d_states[idx]);*/
        static int id = 0;
        if (id >= N) {
            printf("Run out of random values!");			
        }
		return d_RandomArray[id++];
    }
}