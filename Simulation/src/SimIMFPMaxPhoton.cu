#include "hip/hip_runtime.h"
#include "SimIMFPMaxPhoton.hh"

#include <cstdio>
#include <iostream>
#include "Utils.h"  

namespace IMFPMaxPhoton {
    hipArray_t array;
    hipTextureObject_t tex;
    __device__ hipTextureObject_t d_tex;
}

void SimIMFPMaxPhoton::initializeTable(){
    float aux;
	aux = (float)fEmin;
    hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::Emin), &aux, sizeof(int));
	aux = (float)fEmax;
	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::Emax), &aux, sizeof(int));
	aux = (float)fData.GetInvDelta();
	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::InvDelta), &aux, sizeof(int));

	std::vector<float> DataY;
	for (int i = 0; i < fData.GetNumData(); ++i) {
		DataY[i] = (float)fData.GetData(i).fY;
	}

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.normalizedCoords = 0;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeClamp;   
	int NumData = fData.GetNumData();
    initCudaTexture(DataY.data(), &NumData, 1, &texDesc, IMFPMaxPhoton::tex, IMFPMaxPhoton::array);

	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::d_tex), &IMFPMaxPhoton::tex, sizeof(hipTextureObject_t));
}

void  SimIMFPMaxPhoton::LoadData(const std::string& dataDir, int verbose) {
  char name[512];
  sprintf(name, "%s/imfp_globalMax.dat", dataDir.c_str());
  FILE* f = fopen(name, "r");
  if (!f) {
    std::cerr << " *** ERROR SimIMFPMaxPhoton::LoadData: \n"
              << "     file = " << name << " not found! "
              << std::endl;
    exit(EXIT_FAILURE);
  }
  // first 3 lines are comments
  for (int i=0; i<3; ++i) { fgets(name, sizeof(name), f); }
  // load the size of the electron energy grid
  int numData;
  fscanf(f, "%d\n", &numData);
  if (verbose >0) {
    std::cout << " == Loading global max of total IMFP data: "
              << numData << " discrete values for Linear interpolation. "
              << std::endl;
  }
  // one additional line of comment
  fgets(name, sizeof(name), f);
  // load the fNumData E, tota-IMFP data and fill in the linear interplator
  fData.SetSize(numData);
  for (int i=0; i<numData; ++i) {
    double ekin, val;
    fscanf(f, "%lg %lg", &ekin, &val);
    fData.FillData(i, ekin, val);
    if (i==0)         { fEmin = ekin; }
    if (i==numData-1) { fEmax = ekin; }
  }
  fclose(f);

  initializeTable();
}
