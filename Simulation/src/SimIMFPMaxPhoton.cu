#include "hip/hip_runtime.h"
#include "SimIMFPMaxPhoton.hh"

#include <cstdio>
#include <iostream>
#include "Utils.h"  

namespace IMFPMaxPhoton {
    __constant__ float Emin;
    __constant__ float Emax;
    __constant__ float InvDelta;

    hipArray_t array;
    hipTextureObject_t tex;
    __device__ hipTextureObject_t d_tex;

    __device__ float GetValue(float xval) {
        float ilow = (xval - IMFPMaxPhoton::Emin) * IMFPMaxPhoton::InvDelta;
        return tex1D<float>(d_tex, ilow + 0.5f);
    }
}

void SimIMFPMaxPhoton::initializeTable(){
    float aux;
	aux = (float)fEmin;
    hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::Emin), &aux, sizeof(int));
	aux = (float)fEmax;
	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::Emax), &aux, sizeof(int));
	aux = (float)fData.GetInvDelta();
	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::InvDelta), &aux, sizeof(int));

	std::vector<float> DataY;
    DataY.resize(fData.GetNumData());
	for (int i = 0; i < fData.GetNumData(); ++i) {
		DataY[i] = (float)fData.GetData(i).fY;
	}

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.normalizedCoords = 0;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeClamp;   
	int NumData = fData.GetNumData();
    initCudaTexture(DataY.data(), &NumData, 1, &texDesc, IMFPMaxPhoton::tex, IMFPMaxPhoton::array);

	hipMemcpyToSymbol(HIP_SYMBOL(IMFPMaxPhoton::d_tex), &IMFPMaxPhoton::tex, sizeof(hipTextureObject_t));

    CudaCheckError();
}

void  SimIMFPMaxPhoton::LoadData(const std::string& dataDir, int verbose) {
  char name[512];
  sprintf(name, "%s/imfp_globalMax.dat", dataDir.c_str());
  FILE* f = fopen(name, "r");
  if (!f) {
    std::cerr << " *** ERROR SimIMFPMaxPhoton::LoadData: \n"
              << "     file = " << name << " not found! "
              << std::endl;
    exit(EXIT_FAILURE);
  }
  // first 3 lines are comments
  for (int i=0; i<3; ++i) { fgets(name, sizeof(name), f); }
  // load the size of the electron energy grid
  int numData;
  fscanf(f, "%d\n", &numData);
  if (verbose >0) {
    std::cout << " == Loading global max of total IMFP data: "
              << numData << " discrete values for Linear interpolation. "
              << std::endl;
  }
  // one additional line of comment
  fgets(name, sizeof(name), f);
  // load the fNumData E, tota-IMFP data and fill in the linear interplator
  fData.SetSize(numData);
  for (int i=0; i<numData; ++i) {
    double ekin, val;
    fscanf(f, "%lg %lg", &ekin, &val);
    fData.FillData(i, ekin, val);
    if (i==0)         { fEmin = ekin; }
    if (i==numData-1) { fEmax = ekin; }
  }
  fclose(f);

  initializeTable();
}
